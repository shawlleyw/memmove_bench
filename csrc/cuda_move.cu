#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <iostream>

#include <torch/torch.h>
#include <torch/extension.h>
#include <assert.h>
#include <cstring>
#include <string>

#include "cuda_move.h"

template <class T, int CHUNK_SIZE>
__global__ void permute_tokens_kernel(T *d_out, T *d_in, long *mappings, const int hidden_size) {
    constexpr int WARPSIZE = 32;

    int token_id = blockIdx.x;
    int chunk_id = blockIdx.y;
    int num_warps = blockDim.x / WARPSIZE;

    int tid = threadIdx.x;
    int id_in_warp = tid % WARPSIZE;
    int wid = tid / WARPSIZE;

    int p = mappings[token_id];
    if (p == token_id) {
        return;
    }

    int block_base = chunk_id * CHUNK_SIZE;

    // TODO: deal with fp16 and bf16
    half2 *d_in_half2 = (half2 *)(d_in + token_id * hidden_size + block_base);
    half2 *dest_half2 = (half2 *)(d_out + p * hidden_size + block_base);

    int task_per_warp = CHUNK_SIZE / num_warps / 2;
    int warp_base = wid * task_per_warp;

    #pragma unroll
    for (int i = id_in_warp; i < task_per_warp; i += WARPSIZE) {
        dest_half2[warp_base + i] = d_in_half2[warp_base + i];
    }
}

#define LAUNCH_KERNEL_(SIZE) \
do { \
    constexpr int chunk_size = (SIZE); \
    dim3 grid(num_tokens, hidden_size / chunk_size, 1); \
    permute_tokens_kernel<T, chunk_size><<<grid, block>>>(dest, src, mappings, hidden_size); \
} while(0)
    
template <class T>
void _permute_tokens_cuda(T *dest, T *src, long *mappings, int num_tokens, int hidden_size) {
    assert(hidden_size >= 2048 && hidden_size % 2048 == 0);
    constexpr int num_threads = 128;
    dim3 block(num_threads, 1, 1);

    if (num_tokens <= 128) {
        LAUNCH_KERNEL_(512);
    } else if (num_tokens <= 256) {
        LAUNCH_KERNEL_(1024);
    } else {
        LAUNCH_KERNEL_(2048);
    }
}

torch::Tensor permute_tokens_cuda(torch::Tensor tokens, torch::Tensor mappings) {
    assert(tokens.dim() == 2);
    assert(mappings.dim() == 1);
    assert(tokens.size(0) == mappings.size(0));

    int num_tokens = tokens.size(0);
    int hidden_size = tokens.size(1);

    torch::Tensor out = torch::empty_like(tokens);
   
    AT_DISPATCH_REDUCED_FLOATING_TYPES(tokens.scalar_type(), "permute_tokens_cuda", [&] {
        _permute_tokens_cuda<scalar_t>(out.data_ptr<scalar_t>(), tokens.data_ptr<scalar_t>(), mappings.data_ptr<long>(), num_tokens, hidden_size);
    });

    return out;
}